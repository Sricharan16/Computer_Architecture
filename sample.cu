
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 512
#define threads 48
__global__ void MatAdd(int* A, int* B, int* C){
           int i = blockDim.x*blockIdx.x+threadIdx.x;
            int j = blockDim.y*blockIdx.y+threadIdx.y;
            int idx=i*N+j;
           // if((idx<N*N) && i<N && j<N)
           C[idx] = A[idx] + B[idx];
       }


int main(){

int A[N][N];
int B[N][N];
int C[N][N];
const size_t size = sizeof(int) * size_t(N*N);
int (*pA), (*pB), (*pC);
int *h_C;
hipMalloc(&pA, size);
hipMalloc(&pB, size);
hipMalloc(&pC, size);
h_C= new int[size];
for(int i=0;i<N;i++)
{
for(int j=0;j<N;j++)
{
A[i][j]=i+j;
B[i][j]=2*j-1;

C[i][j]=A[i][j]+B[i][j];
}
}
hipMemcpy(pA, A, size, hipMemcpyHostToDevice);
hipMemcpy(pB, B, size, hipMemcpyHostToDevice);
hipMemcpy(pC, C, size, hipMemcpyHostToDevice);

dim3 threadsPerBlock(threads,threads);
dim3 numOfBlocks(N/threads,N/threads);
MatAdd<<<numOfBlocks,threadsPerBlock>>>(pA,pB,pC);

hipMemcpy(h_C, pC, size, hipMemcpyDeviceToHost);

int i, j; printf("C = \n");
for(i=0;i<N;i++){
    for(j=0;j<N;j++){
        printf("%d ", h_C[i*N+j]);
        if(h_C[i*N+j]!=C[i][j])
        {
                printf("ERROR\n");
                exit(0);
        }
        if(i==N-1 && j==N-1)
                printf("running perfectly\n");
    }
    printf("\n");
}

hipFree(pA);
hipFree(pB);
hipFree(pC);

printf("\n");
        return 0;
}



